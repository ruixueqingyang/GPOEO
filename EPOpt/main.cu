#include "hip/hip_runtime.h"
/*******************************************************************************
Copyright(C), 2020-2020, 瑞雪轻飏
     FileName: main.cpp
       Author: 瑞雪轻飏
      Version: 0.01
Creation Date: 20200820
  Description: 测试 CUPTI 能否测量部分 kernel
       Others: 
*******************************************************************************/

#include "EPOpt.h"
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <nvperf_host.h>
#include <nvperf_target.h>
#include <nvperf_cuda_host.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <typeinfo>
#include <time.h>
#include <sys/time.h>

#define NVPW_API_CALL(apiFuncCall)                                             \
do {                                                                           \
    NVPA_Status _status = apiFuncCall;                                         \
    if (_status != NVPA_STATUS_SUCCESS) {                                      \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define CUPTI_API_CALL(apiFuncCall)                                            \
do {                                                                           \
    CUptiResult _status = apiFuncCall;                                         \
    if (_status != CUPTI_SUCCESS) {                                            \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

// #define METRIC_NAME "sm__warps_launched.avg+"
// #define METRIC_NAME "sm__inst_executed.avg.per_cycle_active"
#define METRIC_NAME "sm__inst_executed.max"

ENERGY_PERFORMANCE_OPTIMIZER EPOptDrv;
static void* MeasureMetric(void* Argv);

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // while(true){
    //     if (i < N){
    //         C[i] = A[i] + B[i];
    //         C[i] -= A[i];
    //         C[i] -= B[i];
    //     }
    // }
    if (i < N){
        C[i] = A[i] + B[i];
    }
}

// Device code
__global__ void VecSub(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] - B[i];
}

static void initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}


static void cleanUp(int *h_A, int *h_B, int *h_C, int *h_D, int *d_A, int *d_B, int *d_C, int *d_D)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);
  if (d_D)
    hipFree(d_D);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
  if (h_D)
    free(h_D);
}

static void RunCUDAKernel()
{

    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int *h_A, *h_B, *h_C, *h_D;
    int *d_A, *d_B, *d_C, *d_D;
    int i, sum, diff;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);
    h_D = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, N);
    initVec(h_B, N);
    memset(h_C, 0, size);
    memset(h_D, 0, size);

    // Allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMalloc((void**)&d_D, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n",
            blocksPerGrid, threadsPerBlock);

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    VecSub<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

    // Verify result
    for (i = 0; i < N; ++i) {
        sum = h_A[i] + h_B[i];
        diff = h_A[i] - h_B[i];
        if (h_C[i] != sum || h_D[i] != diff) {
        fprintf(stderr, "error: result verification failed\n");
        exit(-1);
        }
    }

    cleanUp(h_A, h_B, h_C, h_D, d_A, d_B, d_C, d_D);
}

int main(int argc, char* argv[])
{
    int deviceCount, deviceNum;
    char* metricName;
    std::vector<std::string> vecMetricName;
    hipDevice_t cuDevice;
    hipCtx_t cuContext;

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        return -2;
    }

    if (argc > 1)
        deviceNum = atoi(argv[1]);
    else
        deviceNum = 1;
    printf("CUDA Device Number: %d\n", deviceNum);

    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));
    std::cout << "cuContext = 0x" << std::hex << (void*)cuContext << std::dec << std::endl;
    // DRIVER_API_CALL(hipCtxSetCurrent(cuContext));

    // Get the names of the metrics to collect
    if (argc > 2) {
        metricName = strtok(argv[2], ",");
        while(metricName != NULL)
        {
            vecMetricName.push_back(metricName);
            metricName = strtok(NULL, ",");
        }
    }
    else {
        vecMetricName.push_back(METRIC_NAME);
    }

    EPOptDrv.Init(deviceNum, deviceNum, true, true);

    // 这里启动新线程，测量 GPU 性能特征
    pthread_t TID;
    pthread_attr_t attr;
    pthread_attr_init(&attr);
    pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_DETACHED);
    void* Argv[3];

    Argv[0] = (void*)(&vecMetricName);
    Argv[1] = (void*)(&cuContext);
    // Argv[BufIndex][2] = (void*)buf[BufIndex];

    int err = pthread_create(&TID, &attr, MeasureMetric, (void*)Argv);
    if(err != 0) {
        std::cerr << "ERROR: pthread_create() return code: " << err << std::endl;
        exit(1);
    }

    // double SleepTime = 2.0;
    // double StartTimeStamp;
    // struct timeval TimeStruct;

    // std::cout << "sleep(2)" << std::endl;
    // gettimeofday(&TimeStruct,NULL);
    // StartTimeStamp = (double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6;

    // while((double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6 - StartTimeStamp < SleepTime){
    //     gettimeofday(&TimeStruct,NULL);
    // }
    // std::cout << "sleep(2) completes" << std::endl;



    // std::cout << "EPOptDrv.Begin()" << std::endl;
    // EPOptDrv.Begin(vecMetricName);

    // 这里启动 CUDA kernel
    std::cout << "启动 CUDA kernel" << std::endl;
    for(int i = 0; i < 8; i++){
        RunCUDAKernel();
        double SleepTime = 1.0;
        double StartTimeStamp;
        struct timeval TimeStruct;

        std::cout << "kernel sleep(1)" << std::endl;
        gettimeofday(&TimeStruct,NULL);
        StartTimeStamp = (double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6;

        while((double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6 - StartTimeStamp < SleepTime){
            gettimeofday(&TimeStruct,NULL);
        }
        std::cout << "kernel sleep(1) completes" << std::endl;
    }

    // EPOptDrv.End(1);
    // std::cout << "EPOptDrv.End()" << std::endl;




    // DRIVER_API_CALL(hipCtxDestroy(cuContext));
    std::cout << "CUDA kernel completes" << std::endl;

    while(true){}

    return 0;
}

static void* MeasureMetric(void* Argv){

    std::vector<std::string>& vecMetricName = *(((std::vector<std::string>**)Argv)[0]);
    hipCtx_t cuContext = *(  ((hipCtx_t**)Argv)[1]  );

    // std::cout << "开始获取 cuContext" << std::endl;

    // while 直到有 CUDA context 被创建
    // while((void*)cuContext == NULL){
    //     DRIVER_API_CALL(hipCtxGetCurrent(&cuContext));
    //     std::cout << "cuContext = 0x" << std::hex << (void*)cuContext << std::dec << std::endl;
    //     usleep(1000000);
    // }
    std::cout << "cuContext = 0x" << std::hex << (void*)cuContext << std::dec << std::endl;

    // DRIVER_API_CALL(hipInit(0));
    // DRIVER_API_CALL(hipCtxPushCurrent(cuContext));
    DRIVER_API_CALL(hipCtxSetCurrent(cuContext));

    hipCtx_t cuContext1;
    DRIVER_API_CALL(hipCtxGetCurrent(&cuContext1));
    std::cout << "cuContext1 = 0x" << std::hex << (void*)cuContext1 << std::dec << std::endl;
    
    // 延时
    // std::cout << "sleep(2)" << std::endl;
    sleep(2);

    std::cout << "\nEPOptDrv.Begin()" << std::endl;
    EPOptDrv.Begin(vecMetricName);

    // 延时
    // std::cout << "sleep(3)" << std::endl;
    // sleep(3);
    // std::cout << "sleep(3) completes" << std::endl;

    double SleepTime = 5.0;
    double StartTimeStamp;
    struct timeval TimeStruct;

    std::cout << "EPOptDrv sleep(" << SleepTime << ")" << std::endl;
    gettimeofday(&TimeStruct,NULL);
    StartTimeStamp = (double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6;

    while((double)TimeStruct.tv_sec + (double)TimeStruct.tv_usec * 1e-6 - StartTimeStamp < SleepTime){
        gettimeofday(&TimeStruct,NULL);
    }
    std::cout << "EPOptDrv sleep(" << SleepTime << ") completes" << std::endl;

    EPOptDrv.End(1);
    std::cout << "EPOptDrv.End()\n" << std::endl;

    pthread_exit(NULL);
}